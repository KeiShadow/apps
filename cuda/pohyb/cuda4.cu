#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Paralel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage.
// Image transformation from RGB to BW schema.
//
// ***********************************************************************

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include "pic_type.h"
//#include "math.h"

CUDA_Pic cudaColorPic;
CUDA_Pic cudaBWPic;
CUDA_Pic colorPic;
CUDA_Pic bwPic;

// Demo kernel to tranfrom RGB color schema to BW schema
__global__ void kernel_vlneni( CUDA_Pic colorPic, CUDA_Pic bwPic, int posun, int index )
{

    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    if ( y >= colorPic.Size.y ) return;
    if ( x >= colorPic.Size.x ) return;


    

    uchar4 bgr = colorPic.PData[ y * colorPic.Size.x+x];
    //int position = x+index*posun;
    //int finpos = x*bwPic.Size.y + y+position;
    

    //float sinus = sin((float)(x+index)/posun);
    // float position = (x + posun)*bwPic.Size.y+y;
    //float position =  x* bwPic.Size.y + y + (posun + posun*sinus);
    bwPic.PData[y*bwPic.Size.x+(x+index*posun)]= bgr;


}


void run_vlneni(int posun, int index )
{

    hipError_t cerr;
    int block_size = 16;

    dim3 blocks( ( cudaColorPic.Size.x + block_size-1  ) / block_size, ( cudaColorPic.Size.y + block_size-1  ) / block_size );
    dim3 threads( block_size, block_size );

    hipMemset(cudaBWPic.PData, 0, cudaBWPic.Size.x * cudaBWPic.Size.y * sizeof( uchar4 ));

    // Grid creation, size of grid must be greater than image
    kernel_vlneni<<< blocks, threads >>>( cudaColorPic, cudaBWPic, posun, index );

    if ( ( cerr = hipGetLastError() ) != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

    // Copy new image from GPU device
    cerr = hipMemcpy( bwPic.PData, cudaBWPic.PData, bwPic.Size.x * bwPic.Size.y * sizeof( uchar4 ), hipMemcpyDeviceToHost );
    if ( cerr != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

    /*cerr = hipMemcpy( colorPic.PData, cudaColorPic.PData, colorPic.Size.x * colorPic.Size.y * sizeof( uchar4 ), hipMemcpyDeviceToHost );
        if ( cerr != hipSuccess )
            printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );*/


}

void allocate(CUDA_Pic colorPic2, CUDA_Pic bwPic2)
{
    hipError_t cerr;
        // Memory allocation in GPU device
        colorPic = colorPic2;
        bwPic = bwPic2;
        cudaColorPic.Size = colorPic.Size;
        cudaBWPic.Size = bwPic.Size;

        cerr = hipMalloc( &cudaColorPic.PData, cudaColorPic.Size.x * cudaColorPic.Size.y * sizeof( uchar4 ) );
        if ( cerr != hipSuccess )
            printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

        cerr = hipMalloc( &cudaBWPic.PData, cudaBWPic.Size.x * cudaBWPic.Size.y * sizeof( uchar4 ) );
        if ( cerr != hipSuccess )
            printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

       // hipMemset(cudaBWPic.PData, 0, cudaBWPic.Size.x * cudaBWPic.Size.y * sizeof( uchar4 ));

        // Copy color image to GPU device
        cerr = hipMemcpy( cudaColorPic.PData, colorPic.PData, cudaColorPic.Size.x * cudaColorPic.Size.y * sizeof( uchar4 ), hipMemcpyHostToDevice );
        if ( cerr != hipSuccess )
            printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );
}

void deallocate()
{
    // Free memory
        hipFree( cudaColorPic.PData );/*Uvolneni puvodniho obrazku*/
        hipFree( cudaBWPic.PData );/*Uvolneni deformovaneho obrazku*/
}
