// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Paralel Systems
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage
// Multiplication of elements in float array
//
// ***********************************************************************

#include <hip/hip_runtime.h>

#include <stdio.h>

// Demo kernel for array elements multiplication.
// Every thread selects one element and multiply it. 
__global__ void kernel_mult( float *pole, int L, float Mult )
{
	int l = blockDim.x * blockIdx.x + threadIdx.x;
	// if grid is greater then length of array...
	if ( l >= L ) return;

	pole[ l ] *= Mult;
}

void run_mult( float *P, int Length, float Mult )
{
	hipError_t cerr;
	int threads = 128;
	int blocks = ( Length + threads - 1 ) / threads;

	// Memory allocation in GPU device
	float *cudaP;
	cerr = hipMalloc( &cudaP, Length * sizeof( float ) );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );	

	// Copy data from PC to GPU device
	cerr = hipMemcpy( cudaP, P, Length * sizeof( float ), hipMemcpyHostToDevice );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );	

	// Grid creation
	kernel_mult<<< blocks, threads >>>( cudaP, Length, Mult );

	if ( ( cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	// Copy data from GPU device to PC
	cerr = hipMemcpy( P, cudaP, Length * sizeof( float ), hipMemcpyDeviceToHost );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	// Free memory
	hipFree( cudaP );
}